#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20

int row = 480000;
int col = 464;
float *gmat, *gsum;

__global__ void get_average(float *arr, float *sum, int row, int col){
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    // add all row and make average
    for(int k=threadIdx.x; k<row; k+=blockDim.x){
        for(int q = 0; q<col;q++){
            sum[k] += arr[col*k + q];
        }
        sum[k] /= col;
        for(int q = 0; q<col;q++){
            arr[col*k + q] -= sum[k];
        }

    }


}

int main(){
    char *record, *line;
    
    char buffer[400000];
    int i =0;
    int grid = 1;
    int block = 512;

    /*double **arr = (double**)malloc(row * sizeof(double*));
    for(int k=0;k<row;k++)
        arr[k] = (double*)malloc(col * sizeof(double));
    */
    float *arr = (float*)malloc(row *  col * sizeof(float));
    
    float *sum_array = (float*)malloc(row * sizeof(float));

    FILE *fstream = fopen("matrix.csv","r");
    if(fstream == NULL){
        printf("\n file opening failed ");
        return -1;
    }

    printf("Start load csv\n");
    while((line = fgets(buffer, sizeof(buffer),fstream))!= NULL){
        record = strtok(line, ",");
        while(record != NULL){
            float a = atof(record);
            arr[i++] = a;
            record = strtok(NULL, ",");
        }
    }

    printf("Load Finish%lf\n",arr[5]);


    // Create GPU memory
    int err=  hipMalloc((void**)&gmat, row*col*sizeof(float));
    printf("err is %d\n",err);
    hipMalloc((void**)&gsum, row*sizeof(float));
    hipMemcpy(gmat, arr, row*col*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gsum, sum_array, row*sizeof(float), hipMemcpyHostToDevice);
    

    printf("Start calaute\n");
    // Use Cuda kernel
    get_average<<<grid,block>>>(gmat, gsum, row, col);
    
    hipMemcpy(arr, gmat, row*col*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(sum_array, gsum, row*sizeof(float), hipMemcpyDeviceToHost);
    // all row mine average

    
   
    printf("Now write\n");
    FILE *fp;
    fp=fopen("B1.csv","w+");
    for(int i=0;i<row;i++){
        fprintf(fp,"\n%d",i+1);
        for(int j=0;j<col;j++)
            fprintf(fp,",%lf ",arr[j+i*col]);
    }
    return 0;
}
